#include <hip/hip_runtime.h> // Include CUDA runtime API header files


#include <math.h>
#include <iostream>
#include <sstream>
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include "hipblas.h"
#include <vector>
#include <limits>
#define  MAX_SHARE_SIZE 12000
using namespace std;

const int unroll_factor = 4;

__global__ void MVMult(float *matrix, float *vector, float *result, int M, int N, float bias, float factor, int numShared)
{
    __shared__ float cachedVector[MAX_SHARE_SIZE];
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int numRowsPerThread = numShared / blockDim.x;
    int startIndex = threadIdx.x * numRowsPerThread;
    int endIndex = startIndex + numRowsPerThread - 1;
    for (int i = startIndex; i <= endIndex; i++)
    {
        cachedVector[i] = vector[i];
    }
    int numCopied = blockDim.x * numRowsPerThread;

    __syncthreads();
    if (row < M)
    {
        int i = 0;
        for (; i < numCopied && i < N; i++)
        {
            result[row] += matrix[row * N + i] * cachedVector[i];
        }

        for (;i < numCopied && i + 3 < numCopied; i+=4)
        {
            result[row] += matrix[row * N + i] * cachedVector[i];
            result[row] += matrix[row * N + i + 1] * cachedVector[i + 1];
            result[row] += matrix[row * N + i + 2] * cachedVector[i + 2];
            result[row] += matrix[row * N + i + 3] * cachedVector[i + 3];
        }

        for (; i < N; i++)
        {
            result[row] += matrix[row * N + i] * vector[i];
        }

        result[row] *= factor;
        result[row] += bias;
    }
}

__global__ void VVSub(float *vec1, float *vec2, float *res, int N)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    if (index < N && index + 3 < N)
    {
        res[index] = vec1[index] - vec2[index];
        res[index + 1] = vec1[index + 1] - vec2[index + 1];
        res[index + 2] = vec1[index + 2] - vec2[index + 2];
        res[index + 3] = vec1[index + 3] - vec2[index + 3];
    }
}

__global__ void VVSubLeftOver(float *vec1, float *vec2, float *res, int N, int offset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x + offset;
    if (index < N)
    {
        res[index] = vec1[index] - vec2[index];
    }
}

__global__ void VCMult(float *vec, float num, float *res, int N)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    if (index < N && index + 3 < N)
    {
        res[index] -= vec[index] * num;
        res[index + 1] -= vec[index + 1] * num;
        res[index + 2] -= vec[index + 2] * num;
        res[index + 3] -= vec[index + 3] * num;
    }
}

__global__ void VCMultLeftOver(float *vec, float num, float *res, int N, int offset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x + offset;
    if (index < N)
    {
        res[index] -= vec[index] * num;
    }
}

class Regressor
{ // Define Regressor class
private:
    float *theta, *d_x, *d_y, *d_theta; // Declare pointers for theta, x, y, and theta on device
    int m, n;                           // Declare number of training examples and features

    void transpose(float *x_train, float *x_train_transpose)
    {
        for (int i = 0; i < m; i++)
        {
            for (int j = 0; j < n; j++)
            {
                x_train_transpose[j * m + i] = x_train[i * n + j];
            }
        }
    }

public:
    Regressor(int m, int n) // Constructor
    {
        this->m = m;          // Set number of training examples
        this->n = n;          // Set number of features
        theta = new float[n]; // Allocate memory for theta
        for (int i = 0; i < n; i++)
        { // Initialize theta to zero
            theta[i] = 0.0;
        }
    }

    ~Regressor() // Destructor
    {
        delete[] theta;    // Free memory for theta
        hipFree(d_x);     // Free memory for x on device
        hipFree(d_y);     // Free memory for y on device
        hipFree(d_theta); // Free memory for theta on device
    }

    void fit(float *x_train, float *y_train, float alpha, int iterations)
    {
        float *x_train_transpose = new float[n * m];
        transpose(x_train, x_train_transpose);

        float *d_x_train, *d_x_train_transpose,
            *d_y_train, *d_y_pred, *d_diff,
            *d_theta, *d_delta_theta;
        hipMalloc(&d_x_train, m * n * sizeof(float));
        hipMalloc(&d_x_train_transpose, m * n * sizeof(float));
        hipMalloc(&d_y_train, m * sizeof(float));
        hipMalloc(&d_theta, n * sizeof(float));

        // Copy the training data and initial parameters from host to device
        hipMemcpy(d_x_train, x_train, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_x_train_transpose, x_train_transpose, n * m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y_train, y_train, m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_theta, theta, n * sizeof(float), hipMemcpyHostToDevice);

        int block_size = 1024;
        int grid_size = (m + block_size - 1) / block_size;

        float bias = 0;
        for (int i = 0; i < iterations; i++)
        {
            // calculate y_pred
            hipMalloc(&d_y_pred, m * sizeof(float));
            int numShared = m;
            if (numShared > MAX_SHARE_SIZE)
            {
                numShared = MAX_SHARE_SIZE;
            }
            MVMult<<<grid_size, block_size>>>(d_x_train, d_theta, d_y_pred, m, n, bias, 1, numShared);
            
                        
            float *diff = new float[m];
            hipMalloc(&d_diff, m * sizeof(float));

            int vv_block_size = 512;
            int vv_elementsPerBlock = vv_block_size * unroll_factor;
            int vv_grid_size = m / vv_elementsPerBlock;

            int vv_offset = vv_elementsPerBlock * vv_grid_size;

            int vv_leftOver = m - vv_grid_size * vv_elementsPerBlock;
            int vv_leftOver_block_size = 1024;
            int vv_leftOver_grid_size = (vv_leftOver + vv_leftOver_block_size - 1) / vv_leftOver_block_size;
            hipDeviceSynchronize();
            VVSub<<<vv_grid_size, vv_block_size>>>(d_y_pred, d_y_train, d_diff, m);
            VVSubLeftOver<<<vv_leftOver_grid_size, vv_leftOver_block_size>>>(d_y_pred, d_y_train, d_diff, m, vv_offset);
            hipMemcpy(diff, d_diff, m * sizeof(float), hipMemcpyDeviceToHost);

            hipMalloc(&d_delta_theta, n * sizeof(float));
            hipDeviceSynchronize();
            MVMult<<<1, n>>>(d_x_train_transpose, d_diff, d_delta_theta, n, m, 0, 1.0 / m, numShared);

            float sum = 0;
            for (int j = 0; j < n; j++)
            {
                sum += diff[j];
            }
            float delta_bias = sum / m;
            bias -= alpha * delta_bias;

            int vc_block_size = min(n / unroll_factor, 1024);
            int vc_elementsPerBlock = vc_block_size * unroll_factor;
            int vc_grid_size = n / vc_elementsPerBlock;

            int vc_offset = vc_elementsPerBlock * vc_grid_size;

            int vc_leftOver = n - grid_size * vc_elementsPerBlock;
            int vc_leftOver_block_size = min(vc_leftOver, 1024);
            int vc_leftOver_grid_size = (vc_leftOver + vc_leftOver_block_size - 1) / vc_leftOver_block_size;

            
            hipDeviceSynchronize();
            VCMult<<<vc_grid_size, vc_block_size>>>(d_delta_theta, alpha, d_theta, n);
            VCMultLeftOver<<<vc_leftOver_grid_size, vc_leftOver_block_size>>>(d_delta_theta, alpha, d_theta, n, vc_offset);
            hipDeviceSynchronize();

            // clean up memory
            hipFree(d_delta_theta);
            hipFree(d_y_pred);
            hipFree(d_diff);
            delete[] diff;
        }

        // Copy the final parameters from device back to host
        hipMemcpy(theta, d_theta, n * sizeof(float), hipMemcpyDeviceToHost);

        for (int i = 0; i < n; i++)
        {
            cout << theta[i] << ", ";
        }
        cout << endl;
        // clean up memory
        hipFree(d_x_train);
        hipFree(d_x_train_transpose);
        hipFree(d_y_train);
        hipFree(d_theta);
        delete[] x_train_transpose;
    }

    float *predict(float *x_test, int size)
    {
        float *y_pred = new float[size];

        // Calculate the predicted value using the learned parameters
        for (int i = 0; i < size; i++)
        {
            for (int j = 0; j < n; j++)
            {
                y_pred[i] += theta[j] * x_test[i * n + j];
            }
        }

        // Return the predicted values
        return y_pred;
    }
};

float *parseCSV(string fName, int &m, int &n)
{
    ifstream data(fName);
    string line;
    string item;
    vector<vector<float>> dataMatrix;
    getline(data, line); // skip the title
    while (getline(data, line))
    {
        vector<float> dataVec;
        string data;
        stringstream lineStream(line);

        while (getline(lineStream, item, ','))
        {
            dataVec.push_back(stof(item));
        }
        dataMatrix.push_back(dataVec);
    }
    m = dataMatrix.size();
    n = dataMatrix.at(0).size();
    float *a = new float[m * n];

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            a[i * n + j] = dataMatrix[i][j];
        }
    }

    return a;
};

void printMatrix(float *A, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << A[i * n + j] << " ";
        }
        cout << endl;
    }
}
void findStat(float *A, int m, int n, float &min, float &max, float &avg, float &range)
{
    float sum = 0;
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            sum += A[i * n + j];
            if (A[i * n + j] < min)
            {
                min = A[i * n + j];
            }
            if (A[i * n + j] > max)
            {
                max = A[i * n + j];
            }
        }
    }
    avg = sum / (m * n);
    range = max - min;
}

void normalizeAll(float *A, int m, int n)
{
    float range = -1, max = std::numeric_limits<float>::min(),
          min = std::numeric_limits<float>::max(), avg = 0;
    findStat(A, m, n, min, max, avg, range);
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            A[i * n + j] = (A[i * n + j] - min) / range;
        }
    }
}
/**
 void findStat(float* A, int m, int n, int col,float& min, float& max, float& avg, float& range){
    float sum = 0;

    for(int i = 0; i < m; i++){
        sum += A[i * n + col];

        if(A[i * n + col] < min){
            min = A[i * n + col];
        }
        if(A[i * n + col] > max){
            max = A[i * n + col];

        }
    }
    avg = sum / m;
    range = max - min;

}

void normalizeRow(float* A, int col,int m, int n){
    float range = -1, max = std::numeric_limits<float>::min(),
    min = std::numeric_limits<float>::max(),avg = 0;

    findStat(A, m, n, col,min, max,avg,range);
    for(int i = 0; i < m; i++){
        A[i * n + col] = (A[i * n + col] - min) / range;
    }
}

void normalizeAll(float* A, int m, int n){
    for(int i = 0; i < n; i++){
        normalizeRow(A, i, m, n);
    }
}
*/
int main()
{
    int m, n, y_trainM, y_trainN, x_testM, x_testN;

    // example training data
    float *x_train = parseCSV("x_train.csv", m, n);
    normalizeAll(x_train, m, n);
    float *y_train = parseCSV("y_train.csv", y_trainM, y_trainN);

    // example test data
    float *x_test = parseCSV("x_test.csv", x_testM, x_testN);
    normalizeAll(x_test, x_testM, x_testN);

    // train model
    Regressor regressor(m, n);                          // Create a new instance of the Regressor class with m and n
    float alpha = 0.01;                                 // Set the learning rate alpha
    int iterations = 1000;                              // Set the number of training iterations
    regressor.fit(x_train, y_train, alpha, iterations); // Fit the model to the training data

    // test model
    int size = x_testM / n;
    float *y_pred = regressor.predict(x_test, size); // Predict the output for the test data point

    // Print predictions
    for (int i = 0; i < size; i++)
    {
        cout << y_pred[i] << endl;
    }

    delete[] x_test;
    delete[] x_train;
    delete[] y_train;
    delete[] y_pred;
    return 0; // Exit the program
}
