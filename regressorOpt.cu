#include <hip/hip_runtime.h> // Include CUDA runtime API header files


#include <math.h>
#include <iostream>
#include <sstream>
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <vector>
#include <limits>

using namespace std;

__global__ void MVMult(float *matrix, float *vector, float *result, int M, int N, float bias, float factor)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M)
    {
        for (int i = 0; i < N; i++)
        {
            result[row] += matrix[row * N + i] * vector[i];
        }
        result[row] *= factor;
        result[row] += bias;
    }
}

__global__ void VVSub(float *vec1, float *vec2, float *res, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N)
    {
        res[index] = vec1[index] - vec2[index];
    }
}

__global__ void VCMult(float *vec, float num, float *res, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N)
    {
        res[index] -= vec[index] * num;
    }
}

class Regressor
{ // Define Regressor class
private:
    float *theta, *d_x, *d_y, *d_theta; // Declare pointers for theta, x, y, and theta on device
    int m, n;                           // Declare number of training examples and features

    void transpose(float *x_train, float *x_train_transpose)
    {
        for (int i = 0; i < m; i++)
        {
            for (int j = 0; j < n; j++)
            {
                x_train_transpose[j * m + i] = x_train[i * n + j];
            }
        }
    }

public:
    Regressor(int m, int n) // Constructor
    {
        this->m = m;          // Set number of training examples
        this->n = n;          // Set number of features
        theta = new float[n]; // Allocate memory for theta
        for (int i = 0; i < n; i++)
        { // Initialize theta to zero
            theta[i] = 0.0;
        }
    }

    ~Regressor() // Destructor
    {
        delete[] theta;    // Free memory for theta
        hipFree(d_x);     // Free memory for x on device
        hipFree(d_y);     // Free memory for y on device
        hipFree(d_theta); // Free memory for theta on device
    }

    void fit(float *x_train, float *y_train, float alpha, int iterations)
    {
        float *x_train_transpose = new float[n * m];
        transpose(x_train, x_train_transpose);

        float *d_x_train, *d_x_train_transpose,
            *d_y_train, *d_y_pred, *d_diff,
            *d_theta, *d_delta_theta;
        hipMalloc(&d_x_train, m * n * sizeof(float));
        hipMalloc(&d_x_train_transpose, m * n * sizeof(float));
        hipMalloc(&d_y_train, m * sizeof(float));
        hipMalloc(&d_theta, n * sizeof(float));

        // Copy the training data and initial parameters from host to device
        hipMemcpy(d_x_train, x_train, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_x_train_transpose, x_train_transpose, n * m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y_train, y_train, m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_theta, theta, n * sizeof(float), hipMemcpyHostToDevice);

        int block_size = 1024;
        int grid_size = (m + block_size - 1) / block_size;

        float bias = 0;
        for (int i = 0; i < iterations; i++)
        {
            // calculate y_pred
            hipMalloc(&d_y_pred, m * sizeof(float));
            MVMult<<<grid_size, block_size>>>(d_x_train, d_theta, d_y_pred, m, n, bias, 1);

            float *diff = new float[m];
            hipMalloc(&d_diff, m * sizeof(float));
            hipDeviceSynchronize();
            VVSub<<<grid_size, block_size>>>(d_y_pred, d_y_train, d_diff, m);
            hipMemcpy(diff, d_diff, m * sizeof(float), hipMemcpyDeviceToHost);

            hipMalloc(&d_delta_theta, n * sizeof(float));
            hipDeviceSynchronize();
            MVMult<<<1, n>>>(d_x_train_transpose, d_diff, d_delta_theta, n, m, 0, 1.0 / m);

            float sum = 0;
            for (int j = 0; j < n; j++)
            {
                sum += diff[j];
            }
            float delta_bias = sum / m;
            bias -= alpha * delta_bias;

            hipDeviceSynchronize();
            VCMult<<<1, n>>>(d_delta_theta, alpha, d_theta, n);
            hipDeviceSynchronize();

            // clean up memory
            hipFree(d_delta_theta);
            hipFree(d_y_pred);
            hipFree(d_diff);
            delete[] diff;
        }

        // Copy the final parameters from device back to host
        hipMemcpy(theta, d_theta, n * sizeof(float), hipMemcpyDeviceToHost);

        for (int i = 0; i < n; i++)
        {
            cout << theta[i] << ", ";
        }
        cout << endl;
        // clean up memory
        hipFree(d_x_train);
        hipFree(d_x_train_transpose);
        hipFree(d_y_train);
        hipFree(d_theta);
        delete[] x_train_transpose;
    }

    float *predict(float *x_test, int size)
    {
        float *y_pred = new float[size];

        // Calculate the predicted value using the learned parameters
        for (int i = 0; i < size; i++)
        {
            for (int j = 0; j < n; j++)
            {
                y_pred[i] += theta[j] * x_test[i * n + j];
            }
        }

        // Return the predicted values
        return y_pred;
    }
};

float *parseCSV(string fName, int &m, int &n)
{
    ifstream data(fName);
    string line;
    string item;
    vector<vector<float>> dataMatrix;
    getline(data, line); // skip the title
    while (getline(data, line))
    {
        vector<float> dataVec;
        string data;
        stringstream lineStream(line);

        while (getline(lineStream, item, ','))
        {
            dataVec.push_back(stof(item));
        }
        dataMatrix.push_back(dataVec);
    }
    m = dataMatrix.size();
    n = dataMatrix.at(0).size();
    float *a = new float[m * n];

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            a[i * n + j] = dataMatrix[i][j];
        }
    }

    return a;
};

void printMatrix(float *A, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << A[i * n + j] << " ";
        }
        cout << endl;
    }
}
void findStat(float *A, int m, int n, float &min, float &max, float &avg, float &range)
{
    float sum = 0;
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            sum += A[i * n + j];
            if (A[i * n + j] < min)
            {
                min = A[i * n + j];
            }
            if (A[i * n + j] > max)
            {
                max = A[i * n + j];
            }
        }
    }
    avg = sum / (m * n);
    range = max - min;
}

void normalizeAll(float *A, int m, int n)
{
    float range = -1, max = std::numeric_limits<float>::min(),
          min = std::numeric_limits<float>::max(), avg = 0;
          findStat(A, m, n, min, max, avg, range);
          for(int i = 0; i < m; i++){
            for(int j = 0; j < n; j++){
                A[i * n + j] = (A[i * n + j] - min) / range;
            }
          }
}
/**
 void findStat(float* A, int m, int n, int col,float& min, float& max, float& avg, float& range){
    float sum = 0;

    for(int i = 0; i < m; i++){
        sum += A[i * n + col];

        if(A[i * n + col] < min){
            min = A[i * n + col];
        }
        if(A[i * n + col] > max){
            max = A[i * n + col];

        }
    }
    avg = sum / m;
    range = max - min;

}

void normalizeRow(float* A, int col,int m, int n){
    float range = -1, max = std::numeric_limits<float>::min(),
    min = std::numeric_limits<float>::max(),avg = 0;

    findStat(A, m, n, col,min, max,avg,range);
    for(int i = 0; i < m; i++){
        A[i * n + col] = (A[i * n + col] - min) / range;
    }
}

void normalizeAll(float* A, int m, int n){
    for(int i = 0; i < n; i++){
        normalizeRow(A, i, m, n);
    }
}
*/
int main()
{
    int m, n, y_trainM, y_trainN, x_testM, x_testN;

    // example training data
    float *x_train = parseCSV("x_train.csv", m, n);
    normalizeAll(x_train, m, n);
    float *y_train = parseCSV("y_train.csv", y_trainM, y_trainN);

    // example test data
    float *x_test = parseCSV("x_test.csv", x_testM, x_testN);
    normalizeAll(x_test, x_testM, x_testN);

    // train model
    Regressor regressor(m, n);                          // Create a new instance of the Regressor class with m and n
    float alpha = 0.01;                                 // Set the learning rate alpha
    int iterations = 1000;                              // Set the number of training iterations
    regressor.fit(x_train, y_train, alpha, iterations); // Fit the model to the training data

    // test model
    int size = x_testM / n;
    float *y_pred = regressor.predict(x_test, size); // Predict the output for the test data point

    // Print predictions
    for (int i = 0; i < size; i++)
    {
        cout << y_pred[i] << endl;
    }

    delete[] x_test;
    delete[] x_train;
    delete[] y_train;
    delete[] y_pred;
    return 0; // Exit the program
}