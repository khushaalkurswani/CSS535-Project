#include <hip/hip_runtime.h> // Include CUDA runtime API header files


#include <math.h>
#include <iostream>
#include <sstream>
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <vector>
#include <limits>
#include <time.h>

using namespace std;

const int unroll_factor = 4;

__global__ void MVMult(float *matrix, float *vector, float *result, int M, int N, float bias, float factor)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M)
    {
        for (int i = 0; i < N; i++)
        {
            result[row] += matrix[row * N + i] * vector[i];
        }
        result[row] *= factor;
        result[row] += bias;
    }
}

__global__ void VVSub(float *vec1, float *vec2, float *res, int N)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    if (index < N && index + 3 < N)
    {
        res[index] = vec1[index] - vec2[index];
        res[index + 1] = vec1[index + 1] - vec2[index + 1];
        res[index + 2] = vec1[index + 2] - vec2[index + 2];
        res[index + 3] = vec1[index + 3] - vec2[index + 3];
    }
}

__global__ void VVSubLeftOver(float *vec1, float *vec2, float *res, int N, int offset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x + offset;
    if (index < N)
    {
        res[index] = vec1[index] - vec2[index];
    }
}

__global__ void VCMult(float *vec, float num, float *res, int N)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    if (index < N && index + 3 < N)
    {
        res[index] -= vec[index] * num;
        res[index + 1] -= vec[index + 1] * num;
        res[index + 2] -= vec[index + 2] * num;
        res[index + 3] -= vec[index + 3] * num;
    }
}

__global__ void VCMultLeftOver(float *vec, float num, float *res, int N, int offset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x + offset;
    if (index < N)
    {
        res[index] -= vec[index] * num;
    }
}

class Regressor
{ // Define Regressor class
private:
    float *theta, *d_x, *d_y, *d_theta; // Declare pointers for theta, x, y, and theta on device
    int m, n;                           // Declare number of training examples and features

    void transpose(float *x_train, float *x_train_transpose)
    {
        for (int i = 0; i < m; i++)
        {
            for (int j = 0; j < n; j++)
            {
                x_train_transpose[j * m + i] = x_train[i * n + j];
            }
        }
    }

public:
    Regressor(int m, int n) // Constructor
    {
        this->m = m;          // Set number of training examples
        this->n = n;          // Set number of features
        theta = new float[n]; // Allocate memory for theta
        for (int i = 0; i < n; i++)
        { // Initialize theta to zero
            theta[i] = 0.0;
        }
    }

    ~Regressor() // Destructor
    {
        delete[] theta;    // Free memory for theta
        hipFree(d_x);     // Free memory for x on device
        hipFree(d_y);     // Free memory for y on device
        hipFree(d_theta); // Free memory for theta on device
    }

    void fit(float *x_train, float *y_train, float alpha, int iterations)
    {
        float *x_train_transpose = new float[n * m];
        transpose(x_train, x_train_transpose);

        float *d_x_train, *d_x_train_transpose,
            *d_y_train, *d_y_pred, *d_diff,
            *d_theta, *d_delta_theta;
        hipMalloc(&d_x_train, m * n * sizeof(float));
        hipMalloc(&d_x_train_transpose, m * n * sizeof(float));
        hipMalloc(&d_y_train, m * sizeof(float));
        hipMalloc(&d_theta, n * sizeof(float));

        // Copy the training data and initial parameters from host to device
        hipMemcpy(d_x_train, x_train, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_x_train_transpose, x_train_transpose, n * m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y_train, y_train, m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_theta, theta, n * sizeof(float), hipMemcpyHostToDevice);

        int block_size = 512;
        int elementsPerBlock = block_size * unroll_factor;
        int grid_size = m / elementsPerBlock;

        int offset = elementsPerBlock * grid_size;

        int leftOver = m - grid_size * elementsPerBlock;
        int leftOver_block_size = 1024;
        int leftOver_grid_size = (leftOver + leftOver_block_size - 1) / leftOver_block_size;

        float bias = 0;
        for (int i = 0; i < iterations; i++)
        {
            // calculate y_pred
            hipMalloc(&d_y_pred, m * sizeof(float));
            MVMult<<<(m + block_size - 1) / block_size, 1024>>>(d_x_train, d_theta, d_y_pred, m, n, bias, 1);

            float *diff = new float[m];
            hipMalloc(&d_diff, m * sizeof(float));
            hipDeviceSynchronize();
            VVSub<<<grid_size, block_size>>>(d_y_pred, d_y_train, d_diff, m);
            VVSubLeftOver<<<leftOver_grid_size, leftOver_block_size>>>(d_y_pred, d_y_train, d_diff, m, offset);
            hipMemcpy(diff, d_diff, m * sizeof(float), hipMemcpyDeviceToHost);

            hipMalloc(&d_delta_theta, n * sizeof(float));
            hipDeviceSynchronize();
            MVMult<<<1, n>>>(d_x_train_transpose, d_diff, d_delta_theta, n, m, 0, 1.0 / m);

            float sum = 0;
            for (int j = 0; j < n; j++)
            {
                sum += diff[j];
            }
            float delta_bias = sum / m;
            bias -= alpha * delta_bias;

            int vc_block_size = min(n / unroll_factor, 1024);
            int vc_elementsPerBlock = vc_block_size * unroll_factor;
            int vc_grid_size = n / vc_elementsPerBlock;

            int vc_offset = vc_elementsPerBlock * vc_grid_size;

            int vc_leftOver = n - grid_size * elementsPerBlock;
            int vc_leftOver_block_size = min(vc_leftOver, 1024);
            int vc_leftOver_grid_size = (vc_leftOver + vc_leftOver_block_size - 1) / vc_leftOver_block_size;

            
            hipDeviceSynchronize();
            VCMult<<<vc_grid_size, vc_block_size>>>(d_delta_theta, alpha, d_theta, n);
            VCMultLeftOver<<<vc_leftOver_grid_size, vc_leftOver_block_size>>>(d_delta_theta, alpha, d_theta, n, vc_offset);
            hipDeviceSynchronize();

            // clean up memory
            hipFree(d_delta_theta);
            hipFree(d_y_pred);
            hipFree(d_diff);
            delete[] diff;
        }

        // Copy the final parameters from device back to host
        hipMemcpy(theta, d_theta, n * sizeof(float), hipMemcpyDeviceToHost);

        for (int i = 0; i < n; i++)
        {
            cout << theta[i] << ", ";
        }
        cout << endl;
        // clean up memory
        hipFree(d_x_train);
        hipFree(d_x_train_transpose);
        hipFree(d_y_train);
        hipFree(d_theta);
        delete[] x_train_transpose;
    }

    float *predict(float *x_test, int size)
    {
        float *y_pred = new float[size];

        // Calculate the predicted value using the learned parameters
        for (int i = 0; i < size; i++)
        {
            for (int j = 0; j < n; j++)
            {
                y_pred[i] += theta[j] * x_test[i * n + j];
            }
        }

        // Return the predicted values
        return y_pred;
    }
};

float *parseCSV(string fName, int &m, int &n)
{
    ifstream data(fName);
    string line;
    string item;
    vector<vector<float>> dataMatrix;
    getline(data, line); // skip the title
    while (getline(data, line))
    {
        vector<float> dataVec;
        string data;
        stringstream lineStream(line);

        while (getline(lineStream, item, ','))
        {
            dataVec.push_back(stof(item));
        }
        dataMatrix.push_back(dataVec);
    }
    m = dataMatrix.size();
    n = dataMatrix.at(0).size();
    float *a = new float[m * n];

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            a[i * n + j] = dataMatrix[i][j];
        }
    }

    return a;
};

void printMatrix(float *A, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << A[i * n + j] << " ";
        }
        cout << endl;
    }
}
void findStat(float *A, int m, int n, float &min, float &max, float &avg, float &range)
{
    float sum = 0;
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            sum += A[i * n + j];
            if (A[i * n + j] < min)
            {
                min = A[i * n + j];
            }
            if (A[i * n + j] > max)
            {
                max = A[i * n + j];
            }
        }
    }
    avg = sum / (m * n);
    range = max - min;
}

void normalizeAll(float *A, int m, int n)
{
    float range = -1, max = std::numeric_limits<float>::min(),
          min = std::numeric_limits<float>::max(), avg = 0;
          findStat(A, m, n, min, max, avg, range);
          for(int i = 0; i < m; i++){
            for(int j = 0; j < n; j++){
                A[i * n + j] = (A[i * n + j] - min) / range;
            }
          }
}
/**
 void findStat(float* A, int m, int n, int col,float& min, float& max, float& avg, float& range){
    float sum = 0;

    for(int i = 0; i < m; i++){
        sum += A[i * n + col];

        if(A[i * n + col] < min){
            min = A[i * n + col];
        }
        if(A[i * n + col] > max){
            max = A[i * n + col];

        }
    }
    avg = sum / m;
    range = max - min;

}

void normalizeRow(float* A, int col,int m, int n){
    float range = -1, max = std::numeric_limits<float>::min(),
    min = std::numeric_limits<float>::max(),avg = 0;

    findStat(A, m, n, col,min, max,avg,range);
    for(int i = 0; i < m; i++){
        A[i * n + col] = (A[i * n + col] - min) / range;
    }
}

void normalizeAll(float* A, int m, int n){
    for(int i = 0; i < n; i++){
        normalizeRow(A, i, m, n);
    }
}
*/

float calcFLOPS(float elapsed, int m, int n) {
    // matrix vector dot product is 2mn and adding bias is m operations 
    //      (multiply by factor is m operations) --> Unecessary operation for 
    //      this step but part of MVMult kernel
    int FLOP = 2 * m * n + 2 * m; 

    // vector subtraction is m, matrix vector dot product is 2nm, and 
    //      multiply by factor is n operations (adding bias is n 
    //      operations) Unecessary operation for  this step but part of 
    //      MVMult kernel
    FLOP += m + 2 * n * m + n;

    // vector sum is m operations and vector constant multiplication m operations
    //      Reusing vector subtraction result from previous step so not included 
    //      in FLOP calculation for this step
    FLOP += 2 * m;

    // constant multiplication is 1 operation and constant substraction is 1 operation
    FLOP += 2;

    return FLOP / elapsed;
}

int main()
{
    int m, n, y_trainM, y_trainN, x_testM, x_testN;

    // example training data
    float *x_train = parseCSV("x_train.csv", m, n);
    normalizeAll(x_train, m, n);
    float *y_train = parseCSV("y_train.csv", y_trainM, y_trainN);

    // example test data
    float *x_test = parseCSV("x_test.csv", x_testM, x_testN);
    normalizeAll(x_test, x_testM, x_testN);

    // train model
    Regressor regressor(m, n);                          // Create a new instance of the Regressor class with m and n
    float alpha = 0.01;                                 // Set the learning rate alpha
    int iterations = 1000;                              // Set the number of training iterations

    clock_t startTraining = clock(); // start training timer
    regressor.fit(x_train, y_train, alpha, iterations); // Fit the model to the training data
    clock_t endTraining = clock(); // start training timer

    double elapsedTraining = (endTraining - startTraining) / (CLOCKS_PER_SEC / pow(10, 3));
	cout << "Training Time: " << elapsedTraining << " milliseconds" << endl;

    float trainingFLOPS = calcFLOPS(elapsedTraining / 1000, m, n);
    cout << "Training FLOPS: " << trainingFLOPS << " FLOPS" << endl;

    // test model
    int size = x_testM / n;
    float *y_pred = regressor.predict(x_test, size); // Predict the output for the test data point

    // Print predictions
    for (int i = 0; i < size; i++)
    {
        cout << y_pred[i] << endl;
    }

    delete[] x_test;
    delete[] x_train;
    delete[] y_train;
    delete[] y_pred;
    return 0; // Exit the program
}